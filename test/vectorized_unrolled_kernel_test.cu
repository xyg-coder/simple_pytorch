#include <gtest/gtest.h>
#include "cuda/CUDAException.h"
#include "cuda/CUDAStream.h"
#include "cuda/Loops.cuh"
#include "utils/Array.h"
#include <hip/hip_runtime.h>

struct FillFunctor {
  FillFunctor(int v): value(v) {}

  __device__ __forceinline__ int operator() () const {
    return value;
  }
private:
  int value;
};

struct AddFunctor {
  __device__ __forceinline__ int operator() (int a, int b) const {
    return a + b;
  }
};

TEST(KernelTest, vectorized_kernel_test_fill) {
  int n = 100000;
  int *d_a;
  int *h_a = new int[n];

  // this is actually dangerous, because it's possible the stream is switched
  C10_CUDA_CHECK(hipMallocAsync(&d_a, n * sizeof(int), c10::cuda::getCurrentCUDAStream()));

  auto data = c10::Array<decltype(d_a), 1>(d_a);
  c10::cuda::launch_vectorized_kernel(n, FillFunctor(100), data);

  C10_CUDA_CHECK(hipMemcpyAsync(h_a, d_a, n * sizeof(int), hipMemcpyDeviceToHost,
    c10::cuda::getCurrentCUDAStream()));
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++) {  
    EXPECT_EQ(h_a[i], 100);
  }  

  hipFree(d_a);  
  delete[] h_a;
}

TEST(KernelTest, unroll_kernel_test_fill) {
  int n = 100000;
  int *d_a;
  int *h_a = new int[n];

  // this is actually dangerous, because it's possible the stream is switched
  C10_CUDA_CHECK(hipMallocAsync(&d_a, n * sizeof(int), c10::cuda::getCurrentCUDAStream()));

  auto data = c10::Array<decltype(d_a), 1>(d_a);
  c10::cuda::launch_unrolled_kernel(n, FillFunctor(100), data);

  C10_CUDA_CHECK(hipMemcpyAsync(h_a, d_a, n * sizeof(int), hipMemcpyDeviceToHost,
    c10::cuda::getCurrentCUDAStream()));
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++) {  
    EXPECT_EQ(h_a[i], 100);
  }  

  hipFree(d_a);  
  delete[] h_a;
}

TEST(KernelTest, vectorized_kernel_test_add) {
  int n = 100000;
  int *d_a, *d_b, *d_c;
  int *h_sum = new int[n];

  // this is actually dangerous, because it's possible the stream is switched
  C10_CUDA_CHECK(hipMallocAsync(&d_a, n * sizeof(int), c10::cuda::getCurrentCUDAStream()));
  C10_CUDA_CHECK(hipMallocAsync(&d_b, n * sizeof(int), c10::cuda::getCurrentCUDAStream()));
  C10_CUDA_CHECK(hipMallocAsync(&d_c, n * sizeof(int), c10::cuda::getCurrentCUDAStream()));

  auto data = c10::Array<decltype(d_a), 3>();
  data[0] = d_c;
  data[1] = d_b;
  data[2] = d_a;

  c10::cuda::launch_vectorized_kernel(n, FillFunctor(100), c10::Array<decltype(d_a), 1>(d_a));
  c10::cuda::launch_vectorized_kernel(n, FillFunctor(200), c10::Array<decltype(d_a), 1>(d_b));
  c10::cuda::launch_vectorized_kernel(n, AddFunctor(), data);

  C10_CUDA_CHECK(hipMemcpyAsync(h_sum, d_c, n * sizeof(int), hipMemcpyDeviceToHost,
    c10::cuda::getCurrentCUDAStream()));
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++) {  
    EXPECT_EQ(h_sum[i], 300);
  }  

  hipFree(d_a);  
  hipFree(d_b);  
  hipFree(d_c);  
  delete[] h_sum;
}

TEST(KernelTest, unroll_kernel_test_add) {
  int n = 100000;
  int *d_a, *d_b, *d_c;
  int *h_sum = new int[n];

  // this is actually dangerous, because it's possible the stream is switched
  C10_CUDA_CHECK(hipMallocAsync(&d_a, n * sizeof(int), c10::cuda::getCurrentCUDAStream()));
  C10_CUDA_CHECK(hipMallocAsync(&d_b, n * sizeof(int), c10::cuda::getCurrentCUDAStream()));
  C10_CUDA_CHECK(hipMallocAsync(&d_c, n * sizeof(int), c10::cuda::getCurrentCUDAStream()));

  auto data = c10::Array<decltype(d_a), 3>();
  data[0] = d_c;
  data[1] = d_b;
  data[2] = d_a;

  c10::cuda::launch_unrolled_kernel(n, FillFunctor(100), c10::Array<decltype(d_a), 1>(d_a));
  c10::cuda::launch_unrolled_kernel(n, FillFunctor(200), c10::Array<decltype(d_a), 1>(d_b));
  c10::cuda::launch_unrolled_kernel(n, AddFunctor(), data);

  C10_CUDA_CHECK(hipMemcpyAsync(h_sum, d_c, n * sizeof(int), hipMemcpyDeviceToHost,
    c10::cuda::getCurrentCUDAStream()));
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++) {  
    EXPECT_EQ(h_sum[i], 300);
  }  

  hipFree(d_a);  
  hipFree(d_b);  
  hipFree(d_c);  
  delete[] h_sum;
}
